
#include <hip/hip_runtime.h>
// indicates a function that runs on the device, 
// and is called from the host code
// compile: nvcc hello.cu
// execute: ./a.out

__global__ void mykernel(void) {
}
int main(void) {
		// a call from host code to device code (kernel launch)
		mykernel<<<1,1>>>();
		printf("Hello World!\n");
		return 0;
}